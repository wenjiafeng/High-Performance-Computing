
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#define N 1024

using namespace std;

__global__ void dotproduct(int *a, int *b, int *c)
{
        __shared__ int shareArray[N];
        shareArray[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
        __syncthreads();

        if (0 == threadIdx.x) {
                int sum = 0;
                for (int i = 0; i < N; i++) {
                        sum += shareArray[i];
        }
        *c = sum;
    }
}

int main(int argx, char *argc[])
{
        int *A, *B, *Cone;
        int size=sizeof(int);
        A = (int*)malloc(N*size);
        B = (int*)malloc(N*size);
        Cone = (int*)malloc(size);

        int *devA, *devB, *devC;
        hipMalloc((void**)&devA, N*size);
        hipMalloc((void**)&devB, N*size);
        hipMalloc((void**)&devC, size);

        //setup input values
        for (int i=0; i<N; i++){
                A[i] = 1;
                B[i] = 1;
        }
        printf("A:\n");
        for (int i = 0; i < N; i++) {
                printf("%5d", A[i]);
        }
        printf("\nB:\n");
        for (int i = 0; i < N; i++) {
                printf("%5d", B[i]);
        }
        printf("\n");

        //copy host value to device
        hipMemcpy(devA, A, N*size, hipMemcpyHostToDevice);
        hipMemcpy(devB, B, N*size, hipMemcpyHostToDevice);

        dotproduct<<<1,N>>>(devA, devB,devC);

        //copy result to host
        hipMemcpy(Cone, devC, size, hipMemcpyDeviceToHost);

        printf("C:\n");
        printf("%5d", *Cone);

        //Free the memory
        hipFree(devA);
        hipFree(devB);
        hipFree(devC);

        free(A);
        free(B);
        free(Cone);

        return 0;

}
  
